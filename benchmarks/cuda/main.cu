#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <nvm_ctrl.h>
#include <nvm_types.h>
#include <nvm_queue.h>
#include <nvm_util.h>
#include <nvm_admin.h>
#include <nvm_error.h>
#include <nvm_cmd.h>
#include <string>
#include <stdexcept>
#include <vector>
#include <cstdio>
#include <cstdint>
#include <cstring>
#include <fcntl.h>
#include <unistd.h>
#include <sys/mman.h>
#include "ctrl.h"
#include "buffer.h"
#include "settings.h"
#include "event.h"
#include "queue.h"
#include "nvm_parallel_queue.h"
#include "nvm_io.h"
#include "page_cache.h"
#include "util.h"
#include <iostream>
#include <fstream>
#ifdef __DIS_CLUSTER__
#include <sisci_api.h>
#endif

using error = std::runtime_error;
using std::string;


__device__ uint get_smid(void) {

     uint ret;

     asm("mov.u32 %0, %smid;" : "=r"(ret) );

     return ret;

}
uint32_t n_ctrls = 1;
const char* const ctrls_paths[] = {"/dev/libnvm0", "/dev/libnvm1", "/dev/libnvm2", "/dev/libnvm3", "/dev/libnvm4"};


__global__
void new_kernel() {
    printf("in threads\n");
}
__device__ void read_data(page_cache_t* pc, QueuePair* qp, const uint64_t starting_lba, const uint64_t n_blocks, const unsigned long long pc_entry) {
    //uint64_t starting_lba = starting_byte >> qp->block_size_log;
    //uint64_t rem_bytes = starting_byte & qp->block_size_minus_1;
    //uint64_t end_lba = CEIL((starting_byte+num_bytes), qp->block_size);

    //uint16_t n_blocks = CEIL(num_bytes, qp->block_size, qp->block_size_log);



    nvm_cmd_t cmd;
    uint16_t cid = get_cid(&(qp->sq));
    //printf("cid: %u\n", (unsigned int) cid);


    nvm_cmd_header(&cmd, cid, NVM_IO_READ, qp->nvmNamespace);
    uint64_t prp1 = pc->prp1[pc_entry];
    uint64_t prp2 = 0;
    if (pc->prps)
        prp2 = pc->prp2[pc_entry];
    //printf("tid: %llu\tstart_lba: %llu\tn_blocks: %llu\tprp1: %p\n", (unsigned long long) threadIdx.x, (unsigned long long) starting_lba, (unsigned long long) n_blocks, (void*) prp1);
    nvm_cmd_data_ptr(&cmd, prp1, prp2);
    nvm_cmd_rw_blks(&cmd, starting_lba, n_blocks);
    uint16_t sq_pos = sq_enqueue(&qp->sq, &cmd);

    uint32_t cq_pos = cq_poll(&qp->cq, cid);
    sq_dequeue(&qp->sq, sq_pos);
    cq_dequeue(&qp->cq, cq_pos);


    put_cid(&qp->sq, cid);


}

__global__
__launch_bounds__(64, 32)
void access_kernel(Controller* ctrls, page_cache_t* pc,  uint32_t req_size, uint32_t n_reqs, unsigned long long* req_count, uint32_t num_ctrls, uint64_t* assignment) {
    //printf("in threads\n");
    uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t bid = blockIdx.x;
    uint32_t smid = get_smid();

    uint32_t ctrl = bid & (num_ctrls-1);
    uint32_t queue = smid & (ctrls[ctrl].n_qps-1);


    if (tid < n_reqs) {
        uint64_t start_block = (assignment[tid]*req_size) >> ctrls[ctrl].d_qps[queue].block_size_log;
        uint64_t n_blocks = req_size >> ctrls[ctrl].d_qps[queue].block_size_log; /// ctrls[ctrl].ns.lba_data_size;;
       
        //read_data(pc, (ctrls[ctrl].d_qps)+(queue),start_block, n_blocks, tid);
        //__syncthreads();
        //read_data(pc, (ctrls[ctrl].d_qps)+(queue),start_block*2, n_blocks, tid);
        //printf("tid: %llu finished\n", (unsigned long long) tid);

    }

}
__global__
__launch_bounds__(64, 32)
void access_kernel(array_t<uint64_t>* dr, uint64_t n_reqs, unsigned long long* req_count, uint64_t* assignment) {
    //printf("in threads\n");
    uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    //uint32_t bid = blockIdx.x;
    //uint32_t smid = get_smid();

    //uint32_t ctrl = bid & (num_ctrls-1);
    //uint32_t queue = smid & (ctrls[ctrl].n_qps-1);


    if (tid < n_reqs) {
        //req_count += (*dr)[tid];
        req_count += dr->seq_read(tid);
        //uint64_t start_block = (assignment[tid]*req_size) >> ctrls[ctrl].d_qps[queue].block_size_log;
        //uint64_t n_blocks = req_size >> ctrls[ctrl].d_qps[queue].block_size_log; /// ctrls[ctrl].ns.lba_data_size;;

        //read_data(pc, (ctrls[ctrl].d_qps)+(queue),start_block, n_blocks, tid);
        //__syncthreads();
        //read_data(pc, (ctrls[ctrl].d_qps)+(queue),start_block*2, n_blocks, tid);
        //printf("tid: %llu finished\n", (unsigned long long) tid);

    }

}

int main(int argc, char** argv) {

    Settings settings;
    try
    {
        settings.parseArguments(argc, argv);
    }
    catch (const string& e)
    {
        fprintf(stderr, "%s\n", e.c_str());
        fprintf(stderr, "%s\n", Settings::usageString(argv[0]).c_str());
        return 1;
    }


    hipDeviceProp_t properties;
    if (hipGetDeviceProperties(&properties, settings.cudaDevice) != hipSuccess)
    {
        fprintf(stderr, "Failed to get CUDA device properties\n");
        return 1;
    }

    try {
        //Controller ctrl(settings.controllerPath, settings.nvmNamespace, settings.cudaDevice);
        
        cuda_err_chk(hipSetDevice(settings.cudaDevice));
        std::vector<Controller*> ctrls(n_ctrls);
        for (size_t i = 0 ; i < n_ctrls; i++)
            ctrls[i] = new Controller(ctrls_paths[i], settings.nvmNamespace, settings.cudaDevice);

        //auto dma = createDma(ctrl.ctrl, NVM_PAGE_ALIGN(64*1024*10, 1UL << 16), settings.cudaDevice, settings.adapter, settings.segmentId);

        //std::cout << dma.get()->vaddr << std::endl;
        //QueuePair h_qp(ctrl, settings, 1);
        //std::cout << "in main: " << std::hex << h_qp.sq.cid << "raw: " << h_qp.sq.cid<< std::endl;
        //std::memset(&h_qp, 0, sizeof(QueuePair));
        //prepareQueuePair(h_qp, ctrl, settings, 1);
        //const uint32_t ps, const uint64_t np, const uint64_t c_ps, const Settings& settings, const Controller& ctrl)
        //
        /*
        Controller** d_ctrls;
        cuda_err_chk(hipMalloc(&d_ctrls, n_ctrls*sizeof(Controller*)));
        for (size_t i = 0; i < n_ctrls; i++)
            cuda_err_chk(hipMemcpy(d_ctrls+i, &(ctrls[i]->d_ctrl), sizeof(Controller*), hipMemcpyHostToDevice));
        */
        uint64_t b_size = 1;//64;//64;
        uint64_t g_size = 1;//80*16;//80*16;
        uint64_t n_threads = b_size * g_size;


        uint64_t page_size = 512;
        uint64_t n_pages = settings.numChunks;
        uint64_t total_cache_size = (page_size * n_pages);
        //uint64_t n_pages = total_cache_size/page_size;


        page_cache_t h_pc(page_size, n_pages, settings, ctrls[0][0], (uint64_t) 64, ctrls);
        std::cout << "finished creating cache\n";

        //QueuePair* d_qp;
        page_cache_t* d_pc = (page_cache_t*) (h_pc.d_pc_ptr);
        #define TYPE uint64_t
        uint64_t n_elems = (1024ULL)*(1024ULL)*(1024ULL)*(2);
        uint64_t t_size = n_elems * sizeof(TYPE);

        range_t<uint64_t> h_range((uint64_t)0, (uint64_t)n_elems, (uint64_t)0, (uint64_t)(t_size/page_size), (uint64_t)0, (uint64_t)page_size, &h_pc, settings);
        range_t<uint64_t>* d_range = (range_t<uint64_t>*) h_range.d_range_ptr;

        std::vector<range_t<uint64_t>*> vr(1);
        vr[0] = & h_range;
        //(const uint64_t num_elems, const uint64_t disk_start_offset, const std::vector<range_t<T>*>& ranges, Settings& settings)
        array_t<uint64_t> a(n_elems, 0, vr, settings);


        std::cout << "finished creating range\n";

        uint64_t* assignment = (uint64_t*) malloc(n_threads*sizeof(uint64_t));
        for (size_t i = 0; i< n_threads; i++)
            assignment[i] = rand() % (n_elems);

        uint64_t* d_assignment;
        cuda_err_chk(hipMalloc(&d_assignment, n_threads*sizeof(uint64_t)));
        cuda_err_chk(hipMemcpy(d_assignment, assignment,  n_threads*sizeof(uint64_t), hipMemcpyHostToDevice));


        unsigned long long* d_req_count;
        cuda_err_chk(hipMalloc(&d_req_count, sizeof(unsigned long long)));
        cuda_err_chk(hipMemset(d_req_count, 0, sizeof(unsigned long long)));
        std::cout << "atlaunch kernel\n";
        char st[15];
        cuda_err_chk(hipDeviceGetPCIBusId(st, 15, settings.cudaDevice));
        std::cout << st << std::endl;

        Event before;
        //access_kernel<<<g_size, b_size>>>(d_ctrls, d_pc, page_size, n_threads, d_req_count, n_ctrls, d_assignment);
        access_kernel<<<g_size, b_size>>>(a.d_array_ptr, n_threads, d_req_count, d_assignment);
        Event after;
        //new_kernel<<<1,1>>>();
        uint8_t* ret_array = (uint8_t*) malloc(n_pages*page_size);

        cuda_err_chk(hipMemcpy(ret_array, h_pc.base_addr,page_size*n_pages, hipMemcpyDeviceToHost));



        double elapsed = after - before;
        uint64_t ios = g_size*b_size;
        uint64_t data = ios*page_size;
        double iops = ((double)ios)/(elapsed/1000000);
        double bandwidth = (((double)data)/(elapsed/1000000))/(1024ULL*1024ULL*1024ULL);
        std::cout << std::dec << "Elapsed: " << elapsed << "\tIOS: "<< ios << "\tData: " << data << std::endl;
        std::cout << std::dec << "IOPs: " << iops << "\tBandwidth(GB/S): " << bandwidth << std::endl;
        std::cout << std::dec << ctrls[0]->ns.lba_data_size << std::endl;

        std::ofstream ofile("data", std::ios::binary | std::ios::trunc);
        ofile.write((char*)ret_array, data);
        ofile.close();

        for (size_t i = 0 ; i < n_ctrls; i++)
            delete ctrls[i];
        //hexdump(ret_array, n_pages*page_size);
/*
        hipFree(d_qp);
        hipFree(d_pc);
        hipFree(d_req_count);
        free(ret_array);
*/

        std::cout << "END\n";

        std::cout << RAND_MAX << std::endl;

    }
    catch (const error& e) {
        fprintf(stderr, "Unexpected error: %s\n", e.what());
        return 1;
    }



}

/*

struct __align__(64) CmdTime
{
    size_t      size;
    uint64_t    submitTime;
    uint64_t    completeTime;
    uint64_t    moveTime;
};


__host__
std::shared_ptr<CmdTime> createReportingList(size_t numEntries, int device)
{
    auto err = hipSetDevice(device);
    if (err != hipSuccess)
    {
        throw err;
    }

    CmdTime* list = nullptr;
    err = hipMalloc(&list, sizeof(CmdTime) * numEntries);
    if (err != hipSuccess)
    {
        throw err;
    }

    return std::shared_ptr<CmdTime>(list, hipFree);
}


__host__
std::shared_ptr<CmdTime> createReportingList(size_t numEntries)
{
    CmdTime* list = nullptr;

    auto err = hipHostAlloc(&list, sizeof(CmdTime) * numEntries, hipHostMallocDefault);
    if (err != hipSuccess)
    {
        throw err;
    }

    return std::shared_ptr<CmdTime>(list, hipHostFree);
}



__device__
void moveBytes(const void* src, size_t srcOffset, void* dst, size_t dstOffset, size_t size)
{
    const uint16_t numThreads = blockDim.x;
    const uint16_t threadNum = threadIdx.x;

    const ulong4* source = (ulong4*) (((const unsigned char*) src) + srcOffset);
    ulong4* destination = (ulong4*) (((unsigned char*) dst) + dstOffset);

    for (size_t i = 0, n = size / sizeof(ulong4); i < n; i += numThreads)
    {
        destination[i + threadNum] = source[i + threadNum];
    }
}


__device__
void waitForIoCompletion(nvm_queue_t* cq, nvm_queue_t* sq, int* errCode)
{
    const uint16_t numThreads = blockDim.x;

    for (uint16_t i = 0; i < numThreads; ++i)
    {
        nvm_cpl_t* cpl = nullptr;
        while ((cpl = nvm_cq_dequeue(cq)) == nullptr);

        nvm_sq_update(sq);

        if (!NVM_ERR_OK(cpl))
        {
            // *errCount = *errCount + 1;
            *errCode = NVM_ERR_PACK(cpl, 0);
        }
    }

    nvm_cq_update(cq);
}


__device__
nvm_cmd_t* prepareChunk(QueuePair* qp, nvm_cmd_t* last, const uint64_t ioaddr, uint16_t offset, uint64_t blockOffset, uint32_t currChunk)
{
    nvm_cmd_t local;
    const uint16_t numThreads = blockDim.x;
    const uint16_t threadNum = threadIdx.x;
    const uint16_t threadOffset = threadNum + numThreads * offset;

    const uint32_t pageSize = qp->pageSize;
    const uint32_t blockSize = qp->blockSize;
    const uint32_t nvmNamespace = qp->nvmNamespace;
    const uint32_t chunkPages = qp->pagesPerChunk;

    // Calculate offsets
    const uint16_t blocksPerChunk = NVM_PAGE_TO_BLOCK(pageSize, blockSize, chunkPages);
    const uint64_t currBlock = NVM_PAGE_TO_BLOCK(pageSize, blockSize, (currChunk + threadNum) * chunkPages);

    // Prepare PRP list building
    void* prpListPtr = NVM_PTR_OFFSET(qp->prpList, pageSize, threadOffset);
    uint64_t prpListAddr = NVM_ADDR_OFFSET(qp->prpListIoAddr, pageSize, threadOffset);
    nvm_prp_list_t prpList = NVM_PRP_LIST_INIT(prpListPtr, true, pageSize, prpListAddr);

    uint64_t addrs[0x1000 / sizeof(uint64_t)]; // FIXME: This assumes that page size is 4K
    for (uint32_t page = 0; page < chunkPages; ++page)
    {
        addrs[page] = NVM_ADDR_OFFSET(ioaddr, pageSize, chunkPages * threadOffset + page);
    }

    // Enqueue commands
    nvm_cmd_t* cmd = nvm_sq_enqueue_n(&qp->sq, last, numThreads, threadNum);

    // Set command fields
    nvm_cmd_header(&local, threadNum, NVM_IO_READ, nvmNamespace);
    nvm_cmd_data(&local, 1, &prpList, chunkPages, addrs);
    nvm_cmd_rw_blks(&local, currBlock + blockOffset, blocksPerChunk);

    *cmd = local;
    __threadfence();
    return cmd;
}



__global__
void moveKernel(void* src, void* dst, size_t chunkSize)
{
    const uint16_t numThreads = blockDim.x;
    moveBytes(src, 0, dst, 0, chunkSize * numThreads);
}



__host__  inline
void launchMoveKernel(size_t pageSize, void* input, void* src, void* dst, size_t currChunk, const Settings& settings)
{
    const auto numPages = settings.numPages;
    const auto numThreads = settings.numThreads;
    const auto chunkSize = pageSize * numPages;

    void* dstPtr = (void*) (((unsigned char*) dst) + chunkSize * currChunk);
    void* inputPtr = (void*) (((unsigned char*) input) + chunkSize * currChunk);

    hipError_t err = hipMemcpyAsync(src, inputPtr, chunkSize * numThreads, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        throw err;
    }

    moveKernel<<<1, numThreads>>>(src, dstPtr, chunkSize);
}



static double launchMoveKernelLoop(void* fileMap, BufferPtr destination, size_t pageSize, const Settings& settings)
{
    const size_t chunkSize = pageSize * settings.numPages;
    const size_t numThreads = settings.numThreads;
    const size_t totalChunks = settings.numChunks * numThreads;

    const size_t sourceBufferSize = NVM_PAGE_ALIGN(chunkSize * numThreads, 1UL << 16);
    auto source = createBuffer(sourceBufferSize, settings.cudaDevice);

    auto err = hipSetDevice(settings.cudaDevice);
    if (err != hipSuccess)
    {
        throw err;
    }

    Event before;
    for (size_t currChunk = 0; currChunk < totalChunks; currChunk += numThreads)
    {
        launchMoveKernel(pageSize, fileMap, source.get(), destination.get(), currChunk, settings);
    }
    Event after;

    err = hipDeviceSynchronize();
    if (err != hipSuccess)
    {
        throw err;
    }

    return after - before;
}



__global__ static
void readDoubleBuffered(QueuePair* qp, const uint64_t ioaddr, void* src, void* dst, size_t numChunks, uint64_t startBlock, int* errCode, CmdTime* times)
{
    const uint16_t numThreads = blockDim.x;
    const uint16_t threadNum = threadIdx.x;
    const uint32_t pageSize = qp->pageSize;
    const size_t chunkSize = qp->pagesPerChunk * pageSize;
    nvm_queue_t* sq = &qp->sq;

    uint64_t blockOffset = startBlock;

    uint32_t currChunk = 0;
    bool bufferOffset = false;
    uint32_t i = 0;

    nvm_cmd_t* last = prepareChunk(qp, nullptr, ioaddr, bufferOffset, blockOffset, currChunk);

    auto beforeSubmit = clock();
    if (threadNum == 0)
    {
        *errCode = 0;
        nvm_sq_submit(sq);
    }
    __syncthreads();

    while (currChunk + numThreads < numChunks)
    {
        // Prepare in advance next chunk
        last = prepareChunk(qp, last, ioaddr, !bufferOffset, blockOffset, currChunk + numThreads);

        // Consume completions for the previous window
        beforeSubmit = clock();
        if (threadNum == 0)
        {
            waitForIoCompletion(&qp->cq, sq, errCode);
            nvm_sq_submit(sq);
        }
        __syncthreads();
        auto afterSync = clock();

        // Move received chunk
        moveBytes(src, bufferOffset * numThreads * chunkSize, dst, currChunk * chunkSize, chunkSize * numThreads);
        auto afterMove = clock();

        // Record statistics
        if (times != nullptr && threadNum == 0)
        {
            CmdTime* t = &times[i];
            t->size = chunkSize * numThreads;
            t->submitTime = beforeSubmit;
            t->completeTime = afterSync;
            t->moveTime = afterMove;
        }
        __syncthreads();

        // Update position and input buffer
        bufferOffset = !bufferOffset;
        currChunk += numThreads;
        ++i;
    }

    // Wait for final buffer to complete
    if (threadNum == 0)
    {
        waitForIoCompletion(&qp->cq, sq, errCode);
    }
    __syncthreads();
    auto afterSync = clock();

    moveBytes(src, bufferOffset * numThreads * chunkSize, dst, currChunk * chunkSize, chunkSize * numThreads);
    auto afterMove = clock();

    // Record statistics
    if (times != nullptr && threadNum == 0)
    {
        CmdTime* t = &times[i];
        t->size = chunkSize * numThreads;
        t->submitTime = beforeSubmit;
        t->completeTime = afterSync;
        t->moveTime = afterMove;
    }
}



__global__ static
void readSingleBuffered(QueuePair* qp, const uint64_t ioaddr, void* src, void* dst, size_t numChunks, uint64_t startBlock, int* errCode, CmdTime* times)
{
    const uint16_t numThreads = blockDim.x;
    const uint16_t threadNum = threadIdx.x;
    const uint32_t pageSize = qp->pageSize;
    const size_t chunkSize = qp->pagesPerChunk * pageSize;
    nvm_queue_t* sq = &qp->sq;

    uint64_t blockOffset = startBlock;

    uint32_t currChunk = 0;
    uint32_t i = 0;

    nvm_cmd_t* cmd = nullptr;

    if (threadNum == 0)
    {
        *errCode = 0;
    }
    __syncthreads();

    while (currChunk < numChunks)
    {
        // Prepare in advance next chunk
        cmd = prepareChunk(qp, cmd, ioaddr, 0, blockOffset, currChunk);

        // Consume completions for the previous window
        auto beforeSubmit = clock();
        if (threadNum == 0)
        {
            nvm_sq_submit(sq);
            waitForIoCompletion(&qp->cq, sq, errCode);
        }
        __syncthreads();
        auto afterSync = clock();

        // Move received chunk
        moveBytes(src, 0, dst, currChunk * chunkSize, chunkSize * numThreads);
        auto afterMove = clock();

        // Record statistics
        if (times != nullptr && threadNum == 0)
        {
            CmdTime* t = &times[i];
            t->size = chunkSize * numThreads;
            t->submitTime = beforeSubmit;
            t->completeTime = afterSync;
            t->moveTime = afterMove;
        }
        __syncthreads();

        // Update position and input buffer
        currChunk += numThreads;
        ++i;
    }
}


static void printStatistics(const Settings& settings, const hipDeviceProp_t& prop, const std::shared_ptr<CmdTime> gpuTimes)
{
    const size_t numChunks = settings.numChunks;
    auto hostTimes = createReportingList(numChunks);

    auto err = hipMemcpy(hostTimes.get(), gpuTimes.get(), sizeof(CmdTime) * numChunks, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        throw err;
    }

    const auto* times = hostTimes.get();
    const double rate = ((double) prop.clockRate) / 1e3; // GPU's clock frequency in MHz

    fprintf(stdout, "#%9s; %12s; %12s; %12s; %12s; %12s; %12s;\n",
            "size", "disk_lat", "disk_bw", "mem_lat", "mem_bw", "cum_lat", "cum_bw");
    fflush(stdout);
    for (size_t i = 0; i < numChunks; ++i)
    {
        const auto& t = times[i];
        auto diskTime = (t.completeTime - t.submitTime) / rate;
        auto moveTime = (t.moveTime - t.completeTime) / rate;
        auto totalTime = (t.moveTime - t.submitTime) / rate;

        auto diskBw = times[i].size / diskTime;
        auto moveBw = times[i].size / moveTime;
        auto totalBw = times[i].size / totalTime;

        fprintf(stdout, "%10zu; %12.3f; %12.3f; %12.3f; %12.3f; %12.3f; %12.3f;\n",
                t.size, diskTime, diskBw, moveTime, moveBw, totalTime, totalBw);
        fflush(stdout);
    }
}


static double launchNvmKernel(const Controller& ctrl, BufferPtr destination, const Settings& settings, const hipDeviceProp_t& prop)
{
    QueuePair queuePair;
    DmaPtr queueMemory = prepareQueuePair(queuePair, ctrl, settings,1);

    const size_t pageSize = ctrl.info.page_size;
    const size_t chunkSize = pageSize * settings.numPages;
    const size_t totalChunks = settings.numChunks * settings.numThreads;

    // Create input buffer
    const size_t sourceBufferSize = NVM_PAGE_ALIGN((settings.doubleBuffered + 1) * chunkSize * settings.numThreads, 1UL << 16);
    auto source = createDma(ctrl.ctrl, sourceBufferSize, settings.cudaDevice, settings.adapter, settings.segmentId + 1); // vaddr is a dev ptr

    // Set up and prepare queues
    auto deviceQueue = createBuffer(sizeof(QueuePair), settings.cudaDevice);
    auto err = hipMemcpy(deviceQueue.get(), &queuePair, sizeof(QueuePair), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        throw err;
    }

    std::shared_ptr<CmdTime> times;
    if (settings.stats)
    {
        times = createReportingList(settings.numChunks, settings.cudaDevice);
    }

    err = hipSetDevice(settings.cudaDevice);
    if (err != hipSuccess)
    {
        throw err;
    }

    // We want to count number of errors
    int* ec = nullptr;
    err = hipMalloc(&ec, sizeof(int));
    if (err != hipSuccess)
    {
        throw err;
    }

    // Launch kernel
    double elapsed = 0;
    try
    {
        Event before;
        if (settings.doubleBuffered)
        {
            readDoubleBuffered<<<1, settings.numThreads>>>((QueuePair*) deviceQueue.get(), source->ioaddrs[0], source->vaddr, destination.get(), totalChunks, settings.startBlock, ec, times.get());
        }
        else
        {
            readSingleBuffered<<<1, settings.numThreads>>>((QueuePair*) deviceQueue.get(), source->ioaddrs[0], source->vaddr, destination.get(), totalChunks, settings.startBlock, ec, times.get());
        }
        Event after;

        err = hipDeviceSynchronize();
        if (err != hipSuccess)
        {
            throw err;
        }

        elapsed = after - before;
    }
    catch (const hipError_t err)
    {
        hipFree(ec);
        throw err;
    }
    catch (const error& e)
    {
        hipFree(ec);
        throw e;
    }

    // Check error status
    int errorCode = 0;
    hipMemcpy(&errorCode, ec, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(ec);

    if (errorCode != 0)
    {
        fprintf(stderr, "WARNING: There were NVM errors: %s\n", nvm_strerror(errorCode));
    }

    if (settings.stats)
    {
        printStatistics(settings, prop, times);
    }

    return elapsed;
}


static void outputFile(BufferPtr data, size_t size, const char* filename)
{
    auto buffer = createBuffer(size);

    hipError_t err = hipMemcpy(buffer.get(), data.get(), size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        throw error(string("Failed to copy data from destination: ") + hipGetErrorString(err));
    }

    FILE* fp = fopen(filename, "wb");
    fwrite(buffer.get(), 1, size, fp);
    fclose(fp);
}


static int useBlockDevice(const Settings& settings, const hipDeviceProp_t& properties)
{
    int fd = open(settings.blockDevicePath, O_RDONLY);
    if (fd < 0)
    {
        fprintf(stderr, "Failed to open block device: %s\n", strerror(errno));
        return 1;
    }

    const size_t pageSize = sysconf(_SC_PAGESIZE);
    const size_t blockSize = 512; // FIXME: specify this from command line
    const size_t totalChunks = settings.numChunks * settings.numThreads;
    const size_t totalPages = totalChunks * settings.numPages;

    fprintf(stderr, "CUDA device           : %u %s (%s)\n", settings.cudaDevice, properties.name, settings.getDeviceBDF().c_str());
#ifdef __DIS_CLUSTER__
    fprintf(stderr, "CUDA device fdid      : %lx\n", settings.cudaDeviceId);
#endif
    fprintf(stderr, "Controller page size  : %zu B\n", pageSize);
    fprintf(stderr, "Assumed block size    : %zu B\n", blockSize);
    fprintf(stderr, "Number of threads     : %zu\n", settings.numThreads);
    fprintf(stderr, "Chunks per thread     : %zu\n", settings.numChunks);
    fprintf(stderr, "Pages per chunk       : %zu\n", settings.numPages);
    fprintf(stderr, "Total number of pages : %zu\n", totalPages);
    fprintf(stderr, "Double buffering      : %s\n", settings.doubleBuffered ? "yes" : "no");

    void* ptr = mmap(nullptr, totalPages * pageSize, PROT_READ, MAP_FILE | MAP_PRIVATE, fd, settings.startBlock * blockSize);
    if (ptr == nullptr || ptr == MAP_FAILED)
    {
        close(fd);
        fprintf(stderr, "Failed to memory map block device: %s\n", strerror(errno));
        return 1;
    }

    try
    {
        auto outputBuffer = createBuffer(totalPages * pageSize);

        double usecs = launchMoveKernelLoop(ptr, outputBuffer, pageSize, settings);

        fprintf(stderr, "Event time elapsed    : %.3f µs\n", usecs);
        fprintf(stderr, "Estimated bandwidth   : %.3f MiB/s\n", (totalPages * pageSize) / usecs);

        if (settings.output != nullptr)
        {
            outputFile(outputBuffer, totalPages * pageSize, settings.output);
        }
    }
    catch (const hipError_t err)
    {
        munmap(ptr, totalPages * pageSize);
        close(fd);
        fprintf(stderr, "Unexpected CUDA error: %s\n", hipGetErrorString(err));
        return 1;
    }
    catch (const error& e)
    {
        munmap(ptr, totalPages * pageSize);
        close(fd);
        fprintf(stderr, "Unexpected error: %s\n", e.what());
        return 1;
    }

    munmap(ptr, totalPages * pageSize);
    close(fd);
    return 0;
}



int main(int argc, char** argv)
{
    Settings settings;
    try
    {
        settings.parseArguments(argc, argv);
    }
    catch (const string& e)
    {
        fprintf(stderr, "%s\n", e.c_str());
        fprintf(stderr, "%s\n", Settings::usageString(argv[0]).c_str());
        return 1;
    }

#ifdef __DIS_CLUSTER__
    sci_error_t err;
    SCIInitialize(0, &err);
    if (err != SCI_ERR_OK)
    {
        fprintf(stderr, "Failed to initialize SISCI: %s\n", SCIGetErrorString(err));
        return 1;
    }

    sci_desc_t sd;
    SCIOpen(&sd, 0, &err);
    if (err != SCI_ERR_OK)
    {
        fprintf(stderr, "Failed to open SISCI descriptor: %s\n", SCIGetErrorString(err));
        return 1;
    }

    sci_smartio_device_t cudaDev;
    if (settings.cudaDeviceId != 0)
    {
        SCIBorrowDevice(sd, &cudaDev, settings.cudaDeviceId, 0, &err);
        if (err != SCI_ERR_OK)
        {
            fprintf(stderr, "Failed to get SmartIO device reference for CUDA device: %s\n", SCIGetErrorString(err));
            return 1;
        }
    }
    else
    {
        SCIRegisterPCIeRequester(sd, settings.adapter, settings.bus, settings.devfn, SCI_FLAG_PCIE_REQUESTER_GLOBAL, &err);
        if (err != SCI_ERR_OK)
        {
            fprintf(stderr, "Failed to register PCI requester: %s\n", SCIGetErrorString(err));
            SCIClose(sd, 0, &err);
            return 1;
        }
        sleep(1); // FIXME: Hack due to race condition in SmartIO
    }
#endif

    hipDeviceProp_t properties;
    if (hipGetDeviceProperties(&properties, settings.cudaDevice) != hipSuccess)
    {
        fprintf(stderr, "Failed to get CUDA device properties\n");
        return 1;
    }

    if (settings.blockDevicePath != nullptr)
    {
        return useBlockDevice(settings, properties);
    }

    try
    {
#ifdef __DIS_CLUSTER__
        Controller ctrl(settings.controllerId, settings.nvmNamespace, settings.adapter, settings.segmentId++);
#else
        Controller ctrl(settings.controllerPath, settings.nvmNamespace);
#endif
        ctrl.reserveQueues(1);

        const size_t pageSize = ctrl.info.page_size;
        const size_t blockSize = ctrl.ns.lba_data_size;
        const size_t chunkSize = pageSize * settings.numPages;
        const size_t totalChunks = settings.numChunks * settings.numThreads;
        const size_t totalPages = totalChunks * settings.numPages;
        const size_t totalBlocks = NVM_PAGE_TO_BLOCK(pageSize, blockSize, totalPages);

        if (chunkSize > ctrl.info.max_data_size)
        {
            throw error("Chunk size can not be larger than controller data size");
        }
        else if (totalBlocks > ctrl.ns.size)
        {
            throw error("Requesting read size larger than disk size");
        }

        fprintf(stderr, "CUDA device           : %u %s (%s)\n", settings.cudaDevice, properties.name, settings.getDeviceBDF().c_str());
#ifdef __DIS_CLUSTER__
        fprintf(stderr, "CUDA device fdid      : %lx\n", settings.cudaDeviceId);
        fprintf(stderr, "Controller fdid       : %lx\n", settings.controllerId);
#endif
        fprintf(stderr, "Controller page size  : %zu B\n", pageSize);
        fprintf(stderr, "Namespace block size  : %zu B\n", blockSize);
        fprintf(stderr, "Number of threads     : %zu\n", settings.numThreads);
        fprintf(stderr, "Chunks per thread     : %zu\n", settings.numChunks);
        fprintf(stderr, "Pages per chunk       : %zu\n", settings.numPages);
        fprintf(stderr, "Total number of pages : %zu\n", totalPages);
        fprintf(stderr, "Total number of blocks: %zu\n", totalBlocks);
        fprintf(stderr, "Double buffering      : %s\n", settings.doubleBuffered ? "yes" : "no");

        auto outputBuffer = createBuffer(ctrl.info.page_size * totalPages, settings.cudaDevice);

#ifdef __DIS_CLUSTER__
        if (settings.cudaDeviceId != 0)
        {
            nvm_dis_ctrl_map_p2p_device(ctrl.ctrl, cudaDev, nullptr);
        }
#endif

        hipError_t err = hipHostRegister((void*) ctrl.ctrl->mm_ptr, NVM_CTRL_MEM_MINSIZE, hipHostRegisterIoMemory);
        if (err != hipSuccess)
        {
            throw error(string("Unexpected error while mapping IO memory (hipHostRegister): ") + hipGetErrorString(err));
        }

        try
        {
            double usecs = launchNvmKernel(ctrl, outputBuffer, settings, properties);

            fprintf(stderr, "Event time elapsed    : %.3f µs\n", usecs);
            fprintf(stderr, "Estimated bandwidth   : %.3f MiB/s\n", (totalPages * pageSize) / usecs);

            if (settings.output != nullptr)
            {
                outputFile(outputBuffer, totalPages * pageSize, settings.output);
            }
        }
        catch (const error& e)
        {
            hipHostUnregister((void*) ctrl.ctrl->mm_ptr);
            throw e;
        }
        catch (const hipError_t err)
        {
            hipHostUnregister((void*) ctrl.ctrl->mm_ptr);
            throw error(string("Unexpected CUDA error (main): ") + hipGetErrorString(err));
        }
    }
    catch (const error& e)
    {
#ifdef __DIS_CLUSTER__
        if (settings.cudaDeviceId)
        {
            SCIReturnDevice(cudaDev, 0, &err);
        }
        SCIUnregisterPCIeRequester(sd, settings.adapter, settings.bus, settings.devfn, 0, &err);
        SCIClose(sd, 0, &err);
        SCITerminate();
#endif
        fprintf(stderr, "Unexpected error: %s\n", e.what());
        return 1;
    }

#ifdef __DIS_CLUSTER__
    if (settings.cudaDeviceId)
    {
        SCIReturnDevice(cudaDev, 0, &err);
    }
    else
    {
        SCIUnregisterPCIeRequester(sd, settings.adapter, settings.bus, settings.devfn, 0, &err);
    }
    SCIClose(sd, 0, &err);
    SCITerminate();
#endif
    return 0;
}
*/
