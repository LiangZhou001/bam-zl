#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <nvm_ctrl.h>
#include <nvm_types.h>
#include <nvm_queue.h>
#include <nvm_util.h>
#include <nvm_admin.h>
#include <nvm_error.h>
#include <nvm_cmd.h>
#include <string>
#include <stdexcept>
#include <vector>
#include <cstdio>
#include <cstdint>
#include <cstring>
#include <fcntl.h>
#include <unistd.h>
#include <sys/mman.h>
#include <ctrl.h>
#include <buffer.h>
#include "settings.h"
#include <event.h>
#include <queue.h>
#include <nvm_parallel_queue.h>
#include <nvm_io.h>
#include <page_cache.h>
#include <util.h>
#include <iostream>
#include <fstream>
#ifdef __DIS_CLUSTER__
#include <sisci_api.h>
#endif

using error = std::runtime_error;
using std::string;



//uint32_t n_ctrls = 1;
const char* const ctrls_paths[] = {"/dev/libnvm0", "/dev/libnvm1", "/dev/libnvm2", "/dev/libnvm3", "/dev/libnvm4", "/dev/libnvm5", "/dev/libnvm6", "/dev/libnvm7"};


__global__
void new_kernel(ulonglong4* dst, ulonglong4* src, size_t num) {
    warp_memcpy<ulonglong4>(dst, src, num);

}
/*
__device__ void read_data(page_cache_t* pc, QueuePair* qp, const uint64_t starting_lba, const uint64_t n_blocks, const unsigned long long pc_entry) {
    //uint64_t starting_lba = starting_byte >> qp->block_size_log;
    //uint64_t rem_bytes = starting_byte & qp->block_size_minus_1;
    //uint64_t end_lba = CEIL((starting_byte+num_bytes), qp->block_size);

    //uint16_t n_blocks = CEIL(num_bytes, qp->block_size, qp->block_size_log);



    nvm_cmd_t cmd;
    uint16_t cid = get_cid(&(qp->sq));
    //printf("cid: %u\n", (unsigned int) cid);


    nvm_cmd_header(&cmd, cid, NVM_IO_READ, qp->nvmNamespace);
    uint64_t prp1 = pc->prp1[pc_entry];
    uint64_t prp2 = 0;
    if (pc->prps)
        prp2 = pc->prp2[pc_entry];
    //printf("tid: %llu\tstart_lba: %llu\tn_blocks: %llu\tprp1: %p\n", (unsigned long long) threadIdx.x, (unsigned long long) starting_lba, (unsigned long long) n_blocks, (void*) prp1);
    nvm_cmd_data_ptr(&cmd, prp1, prp2);
    nvm_cmd_rw_blks(&cmd, starting_lba, n_blocks);
    uint16_t sq_pos = sq_enqueue(&qp->sq, &cmd);

    uint32_t cq_pos = cq_poll(&qp->cq, cid);
    sq_dequeue(&qp->sq, sq_pos);
    cq_dequeue(&qp->cq, cq_pos);


    put_cid(&qp->sq, cid);


}

*/
__global__
void access_kernel(Controller** ctrls, page_cache_t* pc,  uint32_t req_size, uint32_t n_reqs, unsigned long long* req_count, uint32_t num_ctrls, uint64_t* assignment, uint64_t reqs_per_thread) {
    //printf("in threads\n");
    uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t bid = blockIdx.x;
    uint32_t smid = get_smid();

    uint32_t ctrl = (tid/32) % (num_ctrls);
    uint32_t queue = (tid/32) % (ctrls[ctrl]->n_qps);


    if (tid < n_reqs) {
        uint64_t start_block = (assignment[tid]*req_size) >> ctrls[ctrl]->d_qps[queue].block_size_log;
        //uint64_t start_block = (tid*req_size) >> ctrls[ctrl]->d_qps[queue].block_size_log;
        //start_block = tid;
        uint64_t n_blocks = req_size >> ctrls[ctrl]->d_qps[queue].block_size_log; /// ctrls[ctrl].ns.lba_data_size;;
        //printf("tid: %llu\tstart_block: %llu\tn_blocks: %llu\n", (unsigned long long) tid, (unsigned long long) start_block, (unsigned long long) n_blocks);

        for (size_t i = 0; i < reqs_per_thread; i++)
            read_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid);
        //read_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid);
        //read_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid);
        //read_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid);
        //__syncthreads();
        //read_data(pc, (ctrls[ctrl].d_qps)+(queue),start_block*2, n_blocks, tid);
        //printf("tid: %llu finished\n", (unsigned long long) tid);

    }

}
__global__
void access_kernel(array_t<uint64_t>* dr, uint64_t n_reqs, unsigned long long* req_count, uint64_t* assignment) {
    //printf("in threads\n");
    uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    //uint32_t bid = blockIdx.x;
    //uint32_t smid = get_smid();

    //uint32_t ctrl = bid & (num_ctrls-1);
    //uint32_t queue = smid & (ctrls[ctrl].n_qps-1);


    if (tid < n_reqs) {
        //req_count += (*dr)[tid];
        req_count += dr->seq_read(tid);
        //uint64_t start_block = (assignment[tid]*req_size) >> ctrls[ctrl].d_qps[queue].block_size_log;
        //uint64_t n_blocks = req_size >> ctrls[ctrl].d_qps[queue].block_size_log; /// ctrls[ctrl].ns.lba_data_size;;

        //read_data(pc, (ctrls[ctrl].d_qps)+(queue),start_block, n_blocks, tid);
        //__syncthreads();
        //read_data(pc, (ctrls[ctrl].d_qps)+(queue),start_block*2, n_blocks, tid);
        //printf("tid: %llu finished\n", (unsigned long long) tid);

    }

}

int main(int argc, char** argv) {

    Settings settings;
    try
    {
        settings.parseArguments(argc, argv);
    }
    catch (const string& e)
    {
        fprintf(stderr, "%s\n", e.c_str());
        fprintf(stderr, "%s\n", Settings::usageString(argv[0]).c_str());
        return 1;
    }


    hipDeviceProp_t properties;
    if (hipGetDeviceProperties(&properties, settings.cudaDevice) != hipSuccess)
    {
        fprintf(stderr, "Failed to get CUDA device properties\n");
        return 1;
    }

    try {
        //Controller ctrl(settings.controllerPath, settings.nvmNamespace, settings.cudaDevice);
        
        cuda_err_chk(hipSetDevice(settings.cudaDevice));
        std::vector<Controller*> ctrls(settings.n_ctrls);
        for (size_t i = 0 ; i < settings.n_ctrls; i++)
            ctrls[i] = new Controller(ctrls_paths[i], settings.nvmNamespace, settings.cudaDevice, settings.queueDepth, settings.numQueues);

        //auto dma = createDma(ctrl.ctrl, NVM_PAGE_ALIGN(64*1024*10, 1UL << 16), settings.cudaDevice, settings.adapter, settings.segmentId);

        //std::cout << dma.get()->vaddr << std::endl;
        //QueuePair h_qp(ctrl, settings, 1);
        //std::cout << "in main: " << std::hex << h_qp.sq.cid << "raw: " << h_qp.sq.cid<< std::endl;
        //std::memset(&h_qp, 0, sizeof(QueuePair));
        //prepareQueuePair(h_qp, ctrl, settings, 1);
        //const uint32_t ps, const uint64_t np, const uint64_t c_ps, const Settings& settings, const Controller& ctrl)
        //
        /*
        Controller** d_ctrls;
        cuda_err_chk(hipMalloc(&d_ctrls, n_ctrls*sizeof(Controller*)));
        for (size_t i = 0; i < n_ctrls; i++)
            cuda_err_chk(hipMemcpy(d_ctrls+i, &(ctrls[i]->d_ctrl), sizeof(Controller*), hipMemcpyHostToDevice));
        */
        uint64_t b_size = settings.blkSize;//64;
        uint64_t g_size = (settings.numThreads + b_size - 1)/b_size;//80*16;
        uint64_t n_threads = b_size * g_size;


        uint64_t page_size = settings.pageSize;
        uint64_t n_pages = settings.numPages;
        uint64_t total_cache_size = (page_size * n_pages);
        //uint64_t n_pages = total_cache_size/page_size;


        page_cache_t h_pc(page_size, n_pages, settings.cudaDevice, ctrls[0][0], (uint64_t) 64, ctrls);
        std::cout << "finished creating cache\n";

        //QueuePair* d_qp;
        page_cache_t* d_pc = (page_cache_t*) (h_pc.d_pc_ptr);
        #define TYPE uint64_t
        uint64_t n_elems = (1024ULL)*(1024ULL)*(2);
        uint64_t t_size = n_elems * sizeof(TYPE);

        range_t<uint64_t> h_range((uint64_t)0, (uint64_t)n_elems, (uint64_t)0, (uint64_t)(t_size/page_size), (uint64_t)0, (uint64_t)page_size, &h_pc, settings.cudaDevice);
        range_t<uint64_t>* d_range = (range_t<uint64_t>*) h_range.d_range_ptr;

        std::vector<range_t<uint64_t>*> vr(1);
        vr[0] = & h_range;
        //(const uint64_t num_elems, const uint64_t disk_start_offset, const std::vector<range_t<T>*>& ranges, Settings& settings)
        array_t<uint64_t> a(n_elems, 0, vr, settings.cudaDevice);


        std::cout << "finished creating range\n";

        uint64_t* assignment = (uint64_t*) malloc(n_threads*sizeof(uint64_t));
        for (size_t i = 0; i< n_threads; i++)
            assignment[i] = rand() % (n_elems);

        uint64_t* d_assignment;
        cuda_err_chk(hipMalloc(&d_assignment, n_threads*sizeof(uint64_t)));
        cuda_err_chk(hipMemcpy(d_assignment, assignment,  n_threads*sizeof(uint64_t), hipMemcpyHostToDevice));


        unsigned long long* d_req_count;
        cuda_err_chk(hipMalloc(&d_req_count, sizeof(unsigned long long)));
        cuda_err_chk(hipMemset(d_req_count, 0, sizeof(unsigned long long)));
        std::cout << "atlaunch kernel\n";
        char st[15];
        cuda_err_chk(hipDeviceGetPCIBusId(st, 15, settings.cudaDevice));
        std::cout << st << std::endl;

        Event before;
        access_kernel<<<g_size, b_size>>>(h_pc.d_ctrls, d_pc, page_size, n_threads, d_req_count, settings.n_ctrls, d_assignment, settings.numReqs);
        //access_kernel<<<g_size, b_size>>>(a.d_array_ptr, n_threads, d_req_count, d_assignment);
        Event after;
        //new_kernel<<<1,1>>>();
        uint8_t* ret_array = (uint8_t*) malloc(n_pages*page_size);

        cuda_err_chk(hipMemcpy(ret_array, h_pc.base_addr,page_size*n_pages, hipMemcpyDeviceToHost));



        double elapsed = after - before;
        uint64_t ios = g_size*b_size*settings.numReqs;
        uint64_t data = ios*page_size;
        double iops = ((double)ios)/(elapsed/1000000);
        double bandwidth = (((double)data)/(elapsed/1000000))/(1024ULL*1024ULL*1024ULL);
        std::cout << std::dec << "Elapsed: " << elapsed << "\tIOS: "<< ios << "\tData: " << data << std::endl;
        std::cout << std::dec << "IOPs: " << iops << "\tBandwidth(GB/S): " << bandwidth << std::endl;
        std::cout << std::dec << ctrls[0]->ns.lba_data_size << std::endl;

        std::ofstream ofile("../data", std::ios::binary | std::ios::trunc);
        ofile.write((char*)ret_array, data);
        ofile.close();

        for (size_t i = 0 ; i < settings.n_ctrls; i++)
            delete ctrls[i];
        //hexdump(ret_array, n_pages*page_size);
/*
        hipFree(d_qp);
        hipFree(d_pc);
        hipFree(d_req_count);
        free(ret_array);
*/

        std::cout << "END\n";

        std::cout << RAND_MAX << std::endl;

    }
    catch (const error& e) {
        fprintf(stderr, "Unexpected error: %s\n", e.what());
        return 1;
    }



}
