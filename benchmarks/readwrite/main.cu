#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <nvm_ctrl.h>
#include <nvm_types.h>
#include <nvm_queue.h>
#include <nvm_util.h>
#include <nvm_admin.h>
#include <nvm_error.h>
#include <nvm_cmd.h>
#include <string>
#include <stdexcept>
#include <vector>
#include <cstdio>
#include <cstdint>
#include <cstring>
#include <map>
#include <fcntl.h>
#include <unistd.h>
#include <sys/mman.h>
#include <sys/stat.h>
#include <ctrl.h>
#include <buffer.h>
#include "settings.h"
#include <event.h>
#include <queue.h>
#include <nvm_parallel_queue.h>
#include <nvm_io.h>
#include <page_cache.h>
#include <util.h>
#include <iostream>
#include <fstream>
#ifdef __DIS_CLUSTER__
#include <sisci_api.h>
#endif

using error = std::runtime_error;
using std::string;

// const char* const ctrls_paths[] = {"/dev/libnvm0", "/dev/libnvm1", "/dev/libnvm2", "/dev/libnvm3", "/dev/libnvm4", "/dev/libnvm5", "/dev/libnvm6", "/dev/libnvm7"};
const char* const ctrls_paths[] = {"/dev/libnvm0"};

/*
__device__ void read_data(page_cache_t* pc, QueuePair* qp, const uint64_t starting_lba, const uint64_t n_blocks, const unsigned long long pc_entry) {
    //uint64_t starting_lba = starting_byte >> qp->block_size_log;
    //uint64_t rem_bytes = starting_byte & qp->block_size_minus_1;
    //uint64_t end_lba = CEIL((starting_byte+num_bytes), qp->block_size);

    //uint16_t n_blocks = CEIL(num_bytes, qp->block_size, qp->block_size_log);
 
    nvm_cmd_t cmd;
    uint16_t cid = get_cid(&(qp->sq));
    //printf("cid: %u\n", (unsigned int) cid);
 
    nvm_cmd_header(&cmd, cid, NVM_IO_READ, qp->nvmNamespace);
    uint64_t prp1 = pc->prp1[pc_entry];
    uint64_t prp2 = 0;
    if (pc->prps)
        prp2 = pc->prp2[pc_entry];
    //printf("tid: %llu\tstart_lba: %llu\tn_blocks: %llu\tprp1: %p\n", (unsigned long long) threadIdx.x, (unsigned long long) starting_lba, (unsigned long long) n_blocks, (void*) prp1);
    nvm_cmd_data_ptr(&cmd, prp1, prp2);
    nvm_cmd_rw_blks(&cmd, starting_lba, n_blocks);
    uint16_t sq_pos = sq_enqueue(&qp->sq, &cmd);

    uint32_t cq_pos = cq_poll(&qp->cq, cid);
    sq_dequeue(&qp->sq, sq_pos);
    cq_dequeue(&qp->cq, cq_pos);
 
    put_cid(&qp->sq, cid);
 
}
*/

__global__
void sequential_access_kernel(Controller** ctrls, page_cache_t* pc,  uint32_t req_size, uint32_t n_reqs, //unsigned long long* req_count, 
                                uint32_t num_ctrls, uint64_t reqs_per_thread, uint32_t access_type, uint64_t s_offset){
    //printf("in threads\n");
    uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    // uint32_t bid = blockIdx.x;
    // uint32_t smid = get_smid();

    uint32_t ctrl = (tid/32) % (num_ctrls);
    uint32_t queue = (tid/32) % (ctrls[ctrl]->n_qps);

    for (size_t i = 0; i < pc->n_pages; i = i+n_reqs){
        if (tid < n_reqs) {
            uint64_t start_block = (s_offset + i*n_reqs + tid*req_size) >> ctrls[ctrl]->d_qps[queue].block_size_log ;
            uint64_t pc_idx = (tid+i*n_reqs);
            //uint64_t start_block = (tid*req_size) >> ctrls[ctrl]->d_qps[queue].block_size_log;
            //start_block = tid;
            uint64_t n_blocks = req_size >> ctrls[ctrl]->d_qps[queue].block_size_log; /// ctrls[ctrl].ns.lba_data_size;;
            //printf("tid: %llu\tstart_block: %llu\tn_blocks: %llu\n", (unsigned long long) tid, (unsigned long long) start_block, (unsigned long long) n_blocks);
            // uint8_t opcode;
            // for (size_t i = 0; i < reqs_per_thread; i++) {
                if (access_type == READ) {
                    read_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, pc_idx);
                }
                else {
                    write_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, pc_idx);
                }
            // }
            //read_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid);
            //read_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid);
            //read_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid);
            //__syncthreads();
            //read_data(pc, (ctrls[ctrl].d_qps)+(queue),start_block*2, n_blocks, tid);
            //printf("tid: %llu finished\n", (unsigned long long) tid);
        }
    }
}

/*__global__
void random_access_kernel(Controller** ctrls, page_cache_t* pc,  uint32_t req_size, uint32_t n_reqs, unsigned long long* req_count, uint32_t num_ctrls, uint64_t* assignment, uint64_t reqs_per_thread, uint32_t access_type, uint8_t* access_type_assignment) {
    //printf("in threads\n");
    uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t bid = blockIdx.x;
    uint32_t smid = get_smid();

    uint32_t ctrl = (tid/32) % (num_ctrls);
    uint32_t queue = (tid/32) % (ctrls[ctrl]->n_qps);


    if (tid < n_reqs) {
        uint64_t start_block = (assignment[tid]*req_size) >> ctrls[ctrl]->d_qps[queue].block_size_log;
        //uint64_t start_block = (tid*req_size) >> ctrls[ctrl]->d_qps[queue].block_size_log;
        //start_block = tid;
        uint64_t n_blocks = req_size >> ctrls[ctrl]->d_qps[queue].block_size_log; /// ctrls[ctrl].ns.lba_data_size;;
        //printf("tid: %llu\tstart_block: %llu\tn_blocks: %llu\n", (unsigned long long) tid, (unsigned long long) start_block, (unsigned long long) n_blocks);

        uint8_t opcode;
        for (size_t i = 0; i < reqs_per_thread; i++) {
            if (access_type == MIXED) {
                opcode = access_type_assignment[tid];
                access_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid, opcode);
            }
            else if (access_type == READ) {
                read_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid);

            }
            else {
                write_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid);
            }
        }
        //read_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid);
        //read_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid);
        //read_data(pc, (ctrls[ctrl]->d_qps)+(queue),start_block, n_blocks, tid);
        //__syncthreads();
        //read_data(pc, (ctrls[ctrl].d_qps)+(queue),start_block*2, n_blocks, tid);
        //printf("tid: %llu finished\n", (unsigned long long) tid);

    }

}
*/

int main(int argc, char** argv) {

    Settings settings;
    try
    {
        settings.parseArguments(argc, argv);
    }
    catch (const string& e)
    {
        fprintf(stderr, "%s\n", e.c_str());
        fprintf(stderr, "%s\n", Settings::usageString(argv[0]).c_str());
        return 1;
    }

    hipDeviceProp_t properties;
    if (hipGetDeviceProperties(&properties, settings.cudaDevice) != hipSuccess)
    {
        fprintf(stderr, "Failed to get CUDA device properties\n");
        return 1;
    }

    //move to settings.h file
    const char* input_f;
    if(settings.input == nullptr){
        fprintf(stderr, "Input file required\n");
        return 1;
    }
    else {
        input_f = settings.input; 
        printf("File is : %s\n",input_f);
    }

    try {
        void* map_in;
        int fd_in;
        struct stat sb_in;
        
        if((fd_in = open(input_f, O_RDONLY)) == -1){
            fprintf(stderr, "Input file cannot be opened\n");
            return 1;
        }
        
        fstat(fd_in, &sb_in);
        
        map_in = mmap(NULL, sb_in.st_size, PROT_READ, MAP_SHARED, fd_in, 0);
        
        if((map_in == (void*)-1)){
                fprintf(stderr,"Input file map failed\n",map_in);
                return 1;
        }

        // uint64_t* dummyArray= NULL; 
        // cuda_err_chk(hipHostAlloc(&dummyArray, 1024, hipHostMallocDefault)); 
        // cuda_err_chk(hipMemcpy(dummyArray, map_in+16, 1024, hipMemcpyHostToDevice));

        // printf("value at 0: %llu\n", (uint64_t)dummyArray[0]);
        // fflush(stderr);
        // fflush(stdout);
        
        cuda_err_chk(hipSetDevice(settings.cudaDevice));
        std::vector<Controller*> ctrls(settings.n_ctrls);
        for (size_t i = 0 ; i < settings.n_ctrls; i++)
            ctrls[i] = new Controller(ctrls_paths[i], settings.nvmNamespace, settings.cudaDevice, settings.queueDepth, settings.numQueues);

        // unsigned long long* d_req_count;
        // cuda_err_chk(hipMalloc(&d_req_count, sizeof(unsigned long long)));
        // cuda_err_chk(hipMemset(d_req_count, 0, sizeof(unsigned long long)));
        char st[15];
        cuda_err_chk(hipDeviceGetPCIBusId(st, 15, settings.cudaDevice));
        std::cout << st << std::endl;

        uint64_t b_size = settings.blkSize;//64;
        uint64_t g_size = (settings.numThreads + b_size - 1)/b_size;//80*16;
        uint64_t n_threads = b_size * g_size;

        uint64_t page_size = settings.pageSize;
        uint64_t n_pages = settings.numPages;
        uint64_t total_cache_size = (page_size * n_pages);
        uint64_t n_blocks = settings.numBlks;


        page_cache_t h_pc(page_size, n_pages, settings.cudaDevice, ctrls[0][0], (uint64_t) 64, ctrls);
        std::cout << "finished creating cache\n Total Cache size (MBs):" << (total_cache_size/(1024*1024)) <<std::endl;
        fflush(stderr);
        fflush(stdout);

        //QueuePair* d_qp;
        page_cache_t* d_pc = (page_cache_t*) (h_pc.d_pc_ptr);
        
        uint32_t n_tsteps = ((sb_in.st_size-16)/total_cache_size); //ceil 
        uint64_t n_telem = ((sb_in.st_size-16)/sizeof(uint64_t)); 
        uint64_t s_offset = 0; 

        for (uint32_t cstep =0; cstep < n_tsteps; cstep++) {
                    if(s_offset>(sb_in.st_size-16)) //This cannot happen. 
                        break;

                    uint64_t cpysize = std::min(total_cache_size, (sb_in.st_size-16-s_offset));
                    cuda_err_chk(hipMemcpy(d_pc->base_addr, map_in+s_offset+16, cpysize, hipMemcpyHostToDevice));
                    Event before; 
                    sequential_access_kernel<<<g_size, b_size>>>(h_pc.d_ctrls, d_pc, page_size, n_threads, //d_req_count, 
                                                                    settings.n_ctrls, settings.numReqs, settings.accessType, s_offset);
                    Event after;
                    cuda_err_chk(hipDeviceSynchronize());

                    float completed = 100*(total_cache_size*cstep)/(sb_in.st_size-16);
                    double elapsed = after - before;

                    s_offset = s_offset + cpysize; 

                    std::cout << "Completed:" << completed << "Time:" <<elapsed << std::endl;

                    // uint64_t ios = g_size*b_size*settings.numReqs;
                    // uint64_t data = ios*page_size;
                    // double iops = ((double)ios)/(elapsed/1000000);
                    // double bandwidth = (((double)data)/(elapsed/1000000))/(1024ULL*1024ULL*1024ULL);
                    // std::cout << std::dec << "Elapsed Time: " << elapsed << "\tNumber of Ops: "<< ios << "\tData Size (bytes): " << data << std::endl;
                    // std::cout << std::dec << "Ops/sec: " << iops << "\tEffective Bandwidth(GB/S): " << bandwidth << std::endl;
                    
        }


        
        
        // range_t<uint64_t> h_range((uint64_t)0, (uint64_t)n_elems, (uint64_t)0, (uint64_t)(t_size/page_size), (uint64_t)0, (uint64_t)page_size, &h_pc, settings.cudaDevice);
        // range_t<uint64_t>* d_range = (range_t<uint64_t>*) h_range.d_range_ptr;

        // std::vector<range_t<uint64_t>*> vr(1);
        // vr[0] = & h_range;
        // //(const uint64_t num_elems, const uint64_t disk_start_offset, const std::vector<range_t<T>*>& ranges, Settings& settings)
        // array_t<uint64_t> a(n_elems, 0, vr, settings.cudaDevice);

        //std::cout << "finished creating range\n";

        // uint64_t* assignment;
        // uint64_t* d_assignment;
        // if (settings.random) {
        //     assignment = (uint64_t*) malloc(n_threads*sizeof(uint64_t));
        //     for (size_t i = 0; i< n_threads; i++)
        //         assignment[i] = rand() % (n_blocks);


        //     cuda_err_chk(hipMalloc(&d_assignment, n_threads*sizeof(uint64_t)));
        //     cuda_err_chk(hipMemcpy(d_assignment, assignment,  n_threads*sizeof(uint64_t), hipMemcpyHostToDevice));
        // }
        
        // if (settings.random)
        //     random_access_kernel<<<g_size, b_size>>>(h_pc.d_ctrls, d_pc, page_size, n_threads, d_req_count, settings.n_ctrls, d_assignment, settings.numReqs, settings.accessType, d_access_assignment);
        // else
        
        //new_kernel<<<1,1>>>();
        //uint8_t* ret_array = (uint8_t*) malloc(n_pages*page_size);

        //cuda_err_chk(hipMemcpy(ret_array, h_pc.base_addr,page_size*n_pages, hipMemcpyDeviceToHost));
        //std::cout << std::dec << ctrls[0]->ns.lba_data_size << std::endl;

        //std::ofstream ofile("../data", std::ios::binary | std::ios::trunc);
        //ofile.write((char*)ret_array, data);
        //ofile.close();

        for (size_t i = 0 ; i < settings.n_ctrls; i++)
            delete ctrls[i];
        //hexdump(ret_array, n_pages*page_size);

    }
    catch (const error& e) {
        fprintf(stderr, "Unexpected error: %s\n", e.what());
        return 1;
    }



}
